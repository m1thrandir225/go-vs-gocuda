#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>

#include <math.h>
#include <hip/hip_runtime_api.h>

__global__ void matrix_multiplication(float *a, float *b, float *c, int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        float sum = 0.0f;
        for (int k = 0; k < width; ++k) {
			sum += a[row * width + k] * b[k * width + col];
        }
        c[row * width + col] = sum;
    }
}

// __global__ void matrix_multiplication_tiled(float* a, float* b, float* c, int width, int tileDim) {
//     __shared__ float sA[MAX_TILE_DIM][MAX_TILE_DIM];
//     __shared__ float sB[MAX_TILE_DIM][MAX_TILE_DIM];

//     int tx = threadIdx.x;
//     int ty = threadIdx.y;
//     int row = blockIdx.y * tileDim + ty;
//     int col = blockIdx.x * tileDim + tx;


//     float sum = 0.0f;
//     int numTiles = (width + tileDim - 1) / tileDim;

//     for (int tile = 0; tile < numTiles; ++tile) {
//         int a_load_row = row;
//         int a_load_col = tile * tileDim + tx;
//         if ((a_load_row < width & a_load_col < width & ty < tileDim & tx) < tileDim) {
//             sA[ty][tx] = a[a_load_row * width + a_load_col];
//         }
//         else {
//             sA[ty][tx] = 0.0f;
//         }
//         int b_load_row = tile * tileDim + ty;
//         int b_load_col = col;
//         if ((b_load_row < width & b_load_col < width & ty < tileDim & tx) < tileDim) {
//             sB[ty][tx] = b[b_load_row * width + b_load_col];
//         }
//         else {
//             sB[ty][tx] = 0.0f;
//         }
//         __syncthreads();

//         if (ty < tileDim && tx < tileDim) {
//             for (int k = 0; k < tileDim; ++k) {
//                 if ((tile * tileDim + k) < width) {
//                     sum += sA[ty][k] * sB[k][tx];
//                 }
//             }
//         }
//         __syncthreads();
//     }
//     if (row < width && col < width) {
//         c[row * width + col] = sum;
//     }
// }

extern "C" {
    __declspec(dllexport) void matrix_multiplication_wrapper(float* a, float* b, float* c, int size) {
		int total = size * size;

        float* d_a = NULL, * d_b = NULL, * d_c = NULL;
        int size_bytes = total * sizeof(float);

        // Allocate memory on the GPU
        hipMalloc((void**)&d_a, size_bytes);
        hipMalloc((void**)&d_b, size_bytes);
        hipMalloc((void**)&d_c, size_bytes);
        
        // Copy data from host (CPU) to device (GPU)
        hipMemcpy(d_a, a, size_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size_bytes, hipMemcpyHostToDevice);

        // Define grid and block dimensions
        dim3 threadsPerBlock(16, 16);
        dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x, (size + threadsPerBlock.y - 1) / threadsPerBlock.y);

        // Launch the kernel
        matrix_multiplication<<<numBlocks, threadsPerBlock >>>(d_a, d_b, d_c, size);

        // Copy the result back from device to host
        hipMemcpy(c, d_c, size_bytes, hipMemcpyDeviceToHost);

        // Free GPU memory
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }
}

