#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>

#include <math.h>
#include <hip/hip_runtime_api.h>

__global__ void matrix_multiplication(float *a, float *b, float *c, int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        float sum = 0.0f;
        for (int k = 0; k < width; ++k) {
			sum += a[row * width + k] * b[k * width + col];
        }
        c[row * width + col] = sum;
    }
}


extern "C" {
    __declspec(dllexport) void matrix_multiplication_wrapper(float* a, float* b, float* c, int size) {
		int total = size * size;

        float* d_a = NULL, * d_b = NULL, * d_c = NULL;
        int size_bytes = total * sizeof(float);

        // Allocate memory on the GPU
        hipMalloc((void**)&d_a, size_bytes);
        hipMalloc((void**)&d_b, size_bytes);
        hipMalloc((void**)&d_c, size_bytes);
        
        // Copy data from host (CPU) to device (GPU)
        hipMemcpy(d_a, a, size_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size_bytes, hipMemcpyHostToDevice);

        // Define grid and block dimensions
        dim3 threadsPerBlock(16, 16);
        dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x, (size + threadsPerBlock.y - 1) / threadsPerBlock.y);

        // Launch the kernel
        matrix_multiplication<<<numBlocks, threadsPerBlock >>>(d_a, d_b, d_c, size);

        // Copy the result back from device to host
        hipMemcpy(c, d_c, size_bytes, hipMemcpyDeviceToHost);

        // Free GPU memory
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }
}

