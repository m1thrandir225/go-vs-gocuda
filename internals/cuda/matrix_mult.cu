#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>

#include <math.h>
#include <hip/hip_runtime_api.h>

__global__ void matrix_multiplication(float *a, float *b, float *c, int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        float sum = 0.0f;
        for (int k = 0; k < width; ++k) {
			sum += a[row * width + k] * b[k * width + col];
        }
        c[row * width + col] = sum;
    }
}

__global__ void tiled_matrix_mul(float *a, float *b, float *c, int width) {
    const int TILE_SIZE = 16;
   __shared__ float sA[TILE_SIZE][TILE_SIZE];
   __shared__ float sB[TILE_SIZE][TILE_SIZE];


   int bx = blockIdx.x;
   int by = blockIdx.y;
   int tx = threadIdx.x;
   int ty = threadIdx.y;

   int row = by * TILE_SIZE + ty;
   int col = bx * TILE_SIZE + tx;

   float c_value = 0.0f;

      for (int t = 0; t < (width + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        if (row < width && (t * TILE_SIZE + tx) < width) {
            sA[ty][tx] = a[row * width + (t * TILE_SIZE + tx)];
        } else {
            sA[ty][tx] = 0.0f;
        }

        if ((t * TILE_SIZE + ty) < width && col < width) {
            sB[ty][tx] = b[(t * TILE_SIZE + ty) * width+ col];
        } else {
            sB[ty][tx] = 0.0f;
        }
        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k) {
            c_value += sA[ty][k] * sB[k][tx];
        }

        __syncthreads();
    }

    if (row < width && col < width) {
        c[row * width + col] = c_value;
    }
}


extern "C" {
    __declspec(dllexport) void matrix_multiplication_wrapper(float *a, float *b, float *c, int size) {
		int total = size * size;

        float* d_a = NULL, * d_b = NULL, * d_c = NULL;
        int size_bytes = total * sizeof(float);

        // Allocate memory on the GPU
        hipMalloc((void**)&d_a, size_bytes);
        hipMalloc((void**)&d_b, size_bytes);
        hipMalloc((void**)&d_c, size_bytes);
        
        // Copy data from host (CPU) to device (GPU)
        hipMemcpy(d_a, a, size_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size_bytes, hipMemcpyHostToDevice);

        // Define grid and block dimensions
        dim3 threadsPerBlock(16, 16);
        dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x, (size + threadsPerBlock.y - 1) / threadsPerBlock.y);

        // Launch the kernel
        matrix_multiplication<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, size);

        // Copy the result back from device to host
        hipMemcpy(c, d_c, size_bytes, hipMemcpyDeviceToHost);

        // Free GPU memory
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }

    __declspec(dllexport) void tiled_matrix_multiplication_wrapper(float *a, float *b, float *c, int size) {
        int total = size * size;
        const int TILE_SIZE = 16;

        float *d_a = NULL, *d_b = NULL, *d_c = NULL;
        int size_bytes = total * sizeof(float);

        //Allocate GPU memory
        hipMalloc((void**)&d_a, size_bytes);
        hipMalloc((void**)&d_b, size_bytes);
        hipMalloc((void**)&d_c, size_bytes);

        // Copy data from Host to device
        hipMemcpy(d_a, a, size_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size_bytes, hipMemcpyHostToDevice);

        // Define grid and block dimensions
        int grid_dim_x = (size + TILE_SIZE - 1) / TILE_SIZE;
        int grid_dim_y = (size + TILE_SIZE - 1) / TILE_SIZE;
        dim3 gridDim(grid_dim_x, grid_dim_y);
        dim3 blockDim(TILE_SIZE, TILE_SIZE);

        //Launch kernel
        tiled_matrix_mul<<<gridDim, blockDim>>>(d_a, d_b, d_c, size);

        //Copy result back from device to host
        hipMemcpy(c, d_c, size_bytes, hipMemcpyDeviceToHost);

        //Free memory
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }
}

