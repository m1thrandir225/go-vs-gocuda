#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>

#include <math.h>
#include <hip/hip_runtime_api.h>
#define MAX_TILE_DIM 32

__global__ void matrix_multiplication_tiled(float* a, float* b, float* c, int width, int tileDim) {
    __shared__ float sA[MAX_TILE_DIM][MAX_TILE_DIM];
    __shared__ float sB[MAX_TILE_DIM][MAX_TILE_DIM];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * tileDim + ty;
    int col = blockIdx.x * tileDim + tx;


    float sum = 0.0f;
    int numTiles = (width + tileDim - 1) / tileDim;

    for (int tile = 0; tile < numTiles; ++tile) {
        int a_load_row = row;
        int a_load_col = tile * tileDim + tx;
        if ((a_load_row < width & a_load_col < width & ty < tileDim & tx) < tileDim) {
            sA[ty][tx] = a[a_load_row * width + a_load_col];
        }
        else {
            sA[ty][tx] = 0.0f;
        }
        int b_load_row = tile * tileDim + ty;
        int b_load_col = col;
        if ((b_load_row < width & b_load_col < width & ty < tileDim & tx) < tileDim) {
            sB[ty][tx] = b[b_load_row * width + b_load_col];
        }
        else {
            sB[ty][tx] = 0.0f;
        }
        __syncthreads();

        if (ty < tileDim && tx < tileDim) {
            for (int k = 0; k < tileDim; ++k) {
                if ((tile * tileDim + k) < width) {
                    sum += sA[ty][k] * sB[k][tx];
                }
            }
        }
        __syncthreads();
    }
    if (row < width && col < width) {
        c[row * width + col] = sum;
    }
}

extern "C" {
    __declspec(dllexport) void matrix_multiplication_tiled_wrapper(float *a, float *b, float *c, int size)
    {
        int total = size * size;
        float *d_a = NULL, *d_b = NULL, *d_c = NULL;
        int size_bytes = total * sizeof(float);

        hipMalloc((void**)&d_a, size_bytes);
	    hipMalloc((void**)&d_b, size_bytes);
	    hipMalloc((void**)&d_c, size_bytes);


	    hipMemcpy(d_a, a, size_bytes, hipMemcpyHostToDevice);
	    hipMemcpy(d_b, b, size_bytes, hipMemcpyHostToDevice);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        int tile_dims[] = { 8, 12, 16, 20, 24, 26, 28, 30, 32};
        int num_tile = sizeof(tile_dims) / sizeof(tile_dims[0]);

        float best_tiled_dim_ms = -1.0f;
        int best_tile_dim = 0;

        for (int i = 0; i < num_tile; i++) {
            int current_tile = tile_dims[i];

            if(current_tile > MAX_TILE_DIM) {
                printf("Exceeded max tile dim limit");
                continue;
            }

            printf("Testing dimension: %d\n", current_tile);
            dim3 threadsPerBlock(current_tile, current_tile);

            dim3 numBlocksTiled(( + current_tile - 1) / current_tile, (size + current_tile - 1) / current_tile);
            hipEventRecord(start);
		    matrix_multiplication_tiled<<<numBlocksTiled, threadsPerBlock>>>(d_a, d_b, d_c, size, current_tile);

            
            hipGetLastError();
            hipEventRecord(stop);
            hipEventSynchronize(stop);
		    float current_tiled_time_ms;

		    hipEventElapsedTime(&current_tiled_time_ms, start, stop);

            if(best_tiled_dim_ms < 0 || current_tiled_time_ms < best_tiled_dim_ms) {
                best_tiled_dim_ms = current_tiled_time_ms;
                best_tile_dim = current_tile;
                hipMemcpy(c, d_c, size_bytes, hipMemcpyDeviceToHost);
		    }
		        hipMemset(d_c, 0, size_bytes);
        }
        printf("Best tile dimension: %d with time: %f ms\n", best_tile_dim, best_tiled_dim_ms);

        hipEventDestroy(start);
        hipEventDestroy(stop);
	    hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }
}